#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float4 *pos, *vel; } BodySystem;

void randomizeBodies(float4 *pos, float4 *vel, int n) {
  srand(42);
  for (int i = 0; i < n; i++) {
    pos[i].x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    pos[i].y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    pos[i].z = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    pos[i].w = 0.0f; // Ignored or set to 0
    
    // Initialize velocity (vx, vy, vz), set w to 0
    vel[i].x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    vel[i].y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    vel[i].z = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    vel[i].w = 0.0f; // Ignored or set to 0
  }
}

void savePositionsToFile(BodySystem &p, int nBodies, const char* filepath) {
  FILE *file = fopen(filepath, "w");
  if (file == NULL) {
    printf("Error opening file for writing!\n");
    return;
  }

  for (int i = 0; i < nBodies; i++) {
    // Position from float4
    fprintf(file, "Body %d: Position(%.6f, %.6f, %.6f) ", i, p.pos[i].x, p.pos[i].y, p.pos[i].z);
    // Velocity from float4
    fprintf(file, "Velocity(%.6f, %.6f, %.6f)\n", p.vel[i].x, p.vel[i].y, p.vel[i].z);
  }

  fclose(file);
}

__global__
void bodyForce(float4 *p, float4 *v, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int tile = 0; tile < gridDim.x; tile++) {
      __shared__ float3 spos[BLOCK_SIZE];
      float4 tpos = p[tile * blockDim.x + threadIdx.x];
      spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
      __syncthreads();

      for (int j = 0; j < BLOCK_SIZE; j++) {
        float dx = spos[j].x - p[i].x;
        float dy = spos[j].y - p[i].y;
        float dz = spos[j].z - p[i].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
      }
      __syncthreads();
    }

    v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
  }
}

int main(const int argc, const char** argv) {
  
  int nBodies = 30000;
  int nIters = 10;  // simulation iterations
  if (argc > 1) 
  {
  if(argc < 3)  
  {
  	printf("usage: <executable> <number of N-bodies> <iterations>\n");
  return 0;
  }
  else if (argc > 3)
  {
  	printf("usage: <executable> <number of N-bodies> <iterations>\n");
  return 0;
  }
  
  else
  {
  
  nBodies = atoi(argv[1]);
  nIters = atoi(argv[2]);
  }
}
  else 
  {
  printf("usage: <executable> <number of N-bodies> <iterations>\n");
  return 0;
	}
  
  const float dt = 0.01f; // time step
  
  
  int bytes = 2*nBodies*sizeof(float4);
  float *buf = (float*)malloc(bytes);
  BodySystem p = { (float4*)buf, ((float4*)buf) + nBodies };

  randomizeBodies(p.pos, p.vel, nBodies); // Init pos / vel data

  savePositionsToFile(p, nBodies, "gpu_start.txt");

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  BodySystem d_p = { (float4*)d_buf, ((float4*)d_buf) + nBodies };

  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 

  for (int iter = 1; iter <= nIters; iter++) {
    StartTimer();

    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p.pos, d_p.vel, dt, nBodies);
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p.pos[i].x += p.vel[i].x*dt;
      p.pos[i].y += p.vel[i].y*dt;
      p.pos[i].z += p.vel[i].z*dt;
    }

    const double tElapsed = GetTimer() / 1000.0;
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }

    printf("Iteration %d: %.5f seconds\n", iter, tElapsed);

  }
  double avgTime = totalTime / (double)(nIters-1); 


  
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
  
  savePositionsToFile(p, nBodies, "gpu_end.txt");
  
  free(buf);
  hipFree(d_buf);
  return 0;
}
